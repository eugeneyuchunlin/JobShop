#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>


#define ROW 5000
#define COL 5000


__global__ void matrixAddition(float *a, float *b, float *c, int N){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if( index < N ){
		c[index] = a[index] + b[index];
	}
}

__global__ void test(int ** data){
	int indexx = threadIdx.x;
	int indexy = threadIdx.y;
	// printf("index = %d\n", threadIdx.x);
	// printf("%d\n", data[threadIdx.x][threadIdx.y]);
	data[indexx][indexy] += 1;
}

__global__ void test2(int N){
	int index = threadIdx.x;
	printf("CUDA said: hello world\n");
}


int main(){
	int ** array = (int **)malloc(10*sizeof(int*));
	for(int i = 0; i < 10; ++i){
		array[i] = (int *)malloc(10*sizeof(int));
		for(int j = 0; j < 10; ++j){
			array[i][j] = i + j;	
			printf("%d ", array[i][j]);
		}
		printf("\n");
	}

	

	int ** dev_array;
	hipMalloc((void **)&dev_array, sizeof(int*) *10);
	int * dev_temp;
	int ** dev_temp_array = (int **)malloc(sizeof(int *)*10);
	for(int i = 0; i < 10; ++i){
		hipMalloc((void **)&dev_temp, sizeof(int)*10);
		hipMemcpy(dev_temp, array[i], 10*sizeof(int), hipMemcpyHostToDevice);
		dev_temp_array[i] = dev_temp;
		hipMemcpy(&(dev_array[i]), &dev_temp, sizeof(dev_temp), hipMemcpyHostToDevice);
	}

	dim3 threadsPerBlock(10, 10);
	test<<<1, threadsPerBlock>>>(dev_array);


	int * temp;
	for(int i = 0; i < 10; ++i){
		hipMemcpy(&temp, &(dev_array[i]), sizeof(int *), hipMemcpyDeviceToHost);
		hipMemcpy(array[i], (temp), sizeof(int)*10, hipMemcpyDeviceToHost);
		for(int j = 0; j < 10; ++j){
			printf("%d ", array[i][j]);
		}
		printf("\n");
	}	
	hipDeviceReset();

	return 0;
}
